#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>


// In verion 2, 
// Length of our convolution filter
#define FILTER_LENGTH 7

// Allocate space for the mask in constant memory
__constant__ int filter[MASK_LENGTH];

//
/*
1-D convolution kernel:
array = padded array
result = result array
n = number of elements in the array
*/
__global__ void convolution_1d (int* array, int* mask, int* reuslt, int n, int m){
    //Global thread ID calculation
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the radius of the mask
    int r = m/2;

    // Calculate the starting point for the element
    int start = tid - r;

    //Temp value for calculation
    int temp = 0;

    // Go over each element of the mask
    for (int j = 0; j<m; j++){
        // Ignore elements that hang off
        if(((start + j) >= 0) && (start + j <n)){
            temp += array[start + j] * mask[j];
        }
    }
    //write back the result
    result[tid] = temp;
}

__global__ void convolution_1d_better(int* array, int* reuslt, int n, int m){
    //Global thread ID calculation
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the radius of the mask
    int r = m/2;

    // Calculate the starting point for the element
    int start = tid - r;

    //Temp value for calculation
    int temp = 0;

    // Go over each element of the mask
    for (int j = 0; j<m; j++){
        // Ignore elements that hang off
        if(((start + j) >= 0) && (start + j <n)){
            temp += array[start + j] * filter[j];
        }
    }
    //write back the result
    result[tid] = temp;
}

__global__ void convolution_1d_even_better(int* array, int* reuslt, int n, int m){
    //Global thread ID calculation
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the radius of the mask
    int r = m/2;

    // Calculate the starting point for the element
    int start = tid - r;

    //Temp value for calculation
    int temp = 0;

    // Go over each element of the mask
    for (int j = 0; j<m; j++){
        // Ignore elements that hang off
        if(((start + j) >= 0) && (start + j <n)){
            temp += array[start + j] * filter[j];
        }
    }
    //write back the result
    result[tid] = temp;
}

int main(){

// Number of elements in the result array
int n = 1 << 20; 

// Size of array in bytes
size_t bytes= sizeof(int) * n;

// Numbe rof elements in the convolution filter 
int m = 7;

// Size of filter in bytes
int bytes_m = m * sizeof(int);

// Allocate the array (include edge elements)...
int* h_signal = new int[n];

//Intiialize the input 
for (int i=0;i<n;i++){
    h_signal[i] = rand() % 100;
}

// Filter array
int* h_filter = new int[m];

for (int i=0;i<m;i++){
    h_filter[i] = rand() % 10;
}

// Result array
int* h_result = new int[n];

// Allocate space on the device
int* d_signal, *d_filter, *d_result;
hipMalloc(&d_signal,bytes_n);
hipMalloc(&d_filter,bytes_m);
hipMalloc(&d_result,bytes_n);

//Copy the data to the device
hipMemcpy(d_signal,h_signal,bytes_n,hipMemcpyHostToDevice);

//Version 1
hipMemcpy(d_filter,h_filter,bytes_n,hipMemcpyHostToDevice);
//Version 2
hipMemcpyToSymbol(HIP_SYMBOL(mask),h_filter,bytes_n,)
//Version 3


// Number of threads in the thread block
int BLOCK_SIZE = 256;

// Number of thread blocks
int GRID_SIZE = (n + BLOCK_SIZE -1) / BLOCK_SIZE;

convolution1d <<<GRID_SIZE,BLOCK_SIZE>>>(d_signal,d_filter,d_result,n,m);

hipMemcpy(d_result,h_result,bytes_n,hipMemcpyDeviceToHost);

verify_result(h_a,)

}